#include "hip/hip_runtime.h"
#include "activation.cuh"
#include "error_checking.cuh"

__global__ void ReLuKernel(float* input,
                           const int size);


// Changed
__global__ void SoftmaxKernel(const int output_size,
                                 const int batch_size,
                                 float* values);

void ReLU::operator()(const int output_size,
                      const int batch_size,
                      float* d_value) {
    const int total_size = batch_size * output_size;
    const int threadsPerBlock = 256;
    const int numBlocks = (total_size + threadsPerBlock - 1) / threadsPerBlock;

    ReLuKernel<<<numBlocks, threadsPerBlock>>>(d_value, total_size);
    CHECK_LAST_CUDA_ERROR();
}


// Changed
void SoftMax::operator()(const int batch_size,
    const int output_size,
    float* d_value) {
    const int total_size = batch_size * output_size;
    const int threadsPerBlock = 256;
    const int numBlocks = (total_size + threadsPerBlock - 1) / threadsPerBlock;

    SoftmaxKernel<<<numBlocks, threadsPerBlock>>>(output_size, batch_size, d_value);
    CHECK_LAST_CUDA_ERROR();
}


__global__ void ReLuKernel(float* input,
                           const int output_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < output_size) {
        input[idx] = fmaxf(0, input[idx]);
    }
}


// Changed
__global__ void SoftmaxKernel(const int output_size,
                              const int batch_size,
                              float* values) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= batch_size)
    return;

    // Calculate max for numerical stability (log-sum-exp trick for overflow protection)
    float maxInput = -INFINITY;
    for (int j = 0; j < output_size; ++j) {
        maxInput = fmaxf(maxInput, values[idx * output_size + j]);
    }

    float sum = 0.0f;
    for (int j = 0; j < output_size; ++j) {
        // Subtract maxInput for numerical stability to avoid overflow issues
        sum += expf(values[idx * output_size + j] - maxInput);
    }

    // Calculate Softmax for each element in the batch that this thread should process
    for (int j = 0; j < output_size; ++j) {
        // Apply Softmax formula: exp(input) / sum(exp(input))
        values[idx * output_size + j] = expf(values[idx * output_size + j] - maxInput) / sum;
    }
}